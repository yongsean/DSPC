#include "hip/hip_runtime.h"
#include "split_cuda.h"

/* 
   fills improve, sse, leftsse, and rightsse arrays with values from splitting
   on each column, i.e. each of the arrays is of size numCols()
*/
__global__ void get_split(float **x, float *y, int *col, float *sse, 
                          float *rightSse, float *leftSse, int *method) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    *col = idx;

    // can we pass in function pointers for this part?
    int *data = x[idx];
    switch(method) {
        case 0: // gini
            // do gini split
            break;
        case 1: // anova
            // do anova split
            break;
        default: // error
            break;
    }
}


/* uses a gpu to split on all columns at once. 
 * this function returns a column number. the calling
 * function will need to re-split on the column to
 * get the other data, for the time being
 */
int split(datatable *tab) {
    // get x and y data into array format to be referenced in device function

    // send all columns to device

    // aggregate results and select best split

    // was going to return best column and re-split but if we make 
    // SSE, leftSSE, improve, rightSSE arrays we could just return those
    // and use the correct column
    return 0;
} 
